
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, arraySize>>>(dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; ++i) {
        std::cout << c[i] << " ";
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}

